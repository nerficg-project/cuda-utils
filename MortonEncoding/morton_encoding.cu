#include "hip/hip_runtime.h"
/*
Implementation based on
1. https://github.com/m-schuetz/compute_rasterizer/blob/f2cbb658e6bf58407c385c75d21f3f615f11d5c9/tools/sort_points/Sort_Frugal/src/main.cpp#L79
2. https://gitlab.inria.fr/sibr/sibr_core/-/blob/gaussian_code_release_linux/src/projects/gaussianviewer/renderer/GaussianView.cpp?ref_type=heads#L90
*/

#include "morton_encoding.h"

__device__ __forceinline__ uint64_t splitBy3(uint32_t a) {
	uint64_t x = a & 0x1fffff;
	x = (x | x << 32) & 0x1f00000000ffff;
	x = (x | x << 16) & 0x1f0000ff0000ff;
	x = (x | x << 8) & 0x100f00f00f00f00f;
	x = (x | x << 4) & 0x10c30c30c30c30c3;
	x = (x | x << 2) & 0x1249249249249249;
	return x;
}

__global__ void morton_encode_cu(
        float3 const *const __restrict__ positions,
        float3 const *const __restrict__ minimum_coordinates,
        float const *const __restrict__ cube_size,
        int64_t *const __restrict__ morton_encoding,
        const int n_positions) {
    const int position_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (position_idx >= n_positions) return;
    const float3 position = positions[position_idx];
    const float3 minimum_coordinate = minimum_coordinates[0];
    // could use float instead of double if performance is critical
    const double size = double(cube_size[0]);
    const double normalized_x = double(position.x - minimum_coordinate.x) / size;
    const double normalized_y = double(position.y - minimum_coordinate.y) / size;
    const double normalized_z = double(position.z - minimum_coordinate.z) / size;
    constexpr double factor = 2097151.0; // 2^21 - 1
    const uint32_t x = static_cast<uint32_t>(normalized_x * factor);
    const uint32_t y = static_cast<uint32_t>(normalized_y * factor);
    const uint32_t z = static_cast<uint32_t>(normalized_z * factor);
    const uint64_t morton_code = splitBy3(x) | splitBy3(y) << 1 | splitBy3(z) << 2;
    constexpr int64_t int64_min = -9223372036854775808;
    const int64_t morton_code_torch = static_cast<int64_t>(morton_code) + int64_min;
    morton_encoding[position_idx] = morton_code_torch;
}


at::Tensor morton_encode(
        const at::Tensor& positions,
        const at::Tensor& minimum_coordinates,
        const at::Tensor& cube_size) {
    const int n_positions = positions.size(0);
    at::Tensor morton_encoding = torch::empty({n_positions}, positions.options().dtype(torch::kLong));
    constexpr int block_size = 256;
    const int grid_size = (n_positions + block_size - 1) / block_size;
    morton_encode_cu<<<grid_size, block_size>>>(
        reinterpret_cast<const float3*>(positions.contiguous().data_ptr<float>()),
        reinterpret_cast<const float3*>(minimum_coordinates.contiguous().data_ptr<float>()),
        reinterpret_cast<const float*>(cube_size.contiguous().data_ptr<float>()),
        morton_encoding.data_ptr<int64_t>(),
        n_positions);
    return morton_encoding;
}
